//
// Created by harish on 10.03.21.
//

#include "cudaData.cuh"
#include <iostream>
using namespace std;

int* cudaData::getData(int dataSize,int* hostData) {
    int* deviceVec;
    hipMallocManaged(&deviceVec,dataSize*sizeof(int));
    hipMemcpy(deviceVec, hostData, dataSize*sizeof (int),hipMemcpyHostToDevice);
    return deviceVec;
}

int* cudaData::copyHostToDevice(int* deviceVec, int* hostData, int dataSize){
    hipMemcpy(deviceVec, hostData, dataSize*sizeof (int),hipMemcpyHostToDevice);
    return deviceVec;
}

int* cudaData::allocateGpuMemory(int dataSize, int* hostData){
    int* deviceVec;
    hipMalloc(&deviceVec,dataSize*sizeof(int));
    return deviceVec;
}

int* cudaData::getHostData(int dataSize,int* devData) {
    int* hostData = (int*)malloc(sizeof (int)*dataSize);
    hipMemcpy(hostData,devData, sizeof(int)*dataSize,hipMemcpyDeviceToHost);
    return hostData;
}